#include "libs/cudarender.h"

extern GLuint pbo[2];
extern hipGraphicsResource* cuda_pbo_resource[2];
extern int width;
extern int height;

void createPBOs(int width, int height) {
    glGenBuffers(2, pbo);
    for (int i = 0; i < 2; i++) {
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo[i]);
        glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * 4, nullptr, GL_DYNAMIC_DRAW);
        hipGraphicsGLRegisterBuffer(&cuda_pbo_resource[i], pbo[i], cudaGraphicsMapFlagsWriteDiscard);
    }
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

void updateBuffersize(GLFWwindow *window, int newwidth, int newheight) {
    width = newwidth;
    height = newheight;

    for (int i = 0; i < 2; i++) {
        hipError_t err = hipGraphicsUnregisterResource(cuda_pbo_resource[i]);
        if (err != hipSuccess) {
            std::cerr << "CUDA Error (Unregistering PBO): " << hipGetErrorString(err) << std::endl;
        }
    }
    createPBOs(width, height);
    updateContent(width, height, window);
}

