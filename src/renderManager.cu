#include "hip/hip_runtime.h"
#include "libs/cudarender.h"

extern GLuint pbo[2];
extern hipGraphicsResource* cuda_pbo_resource[2]; 

int pboIndex = 0;

__global__ void fillBuffer(uchar4* buffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        buffer[idx] = make_uchar4(x % 256, y % 256, 128, 255);
    }
}

void updateBuffer(int width, int height) {
    int nextPBO = (pboIndex + 1) % 2;  // Swap buffer
    uchar4* d_buffer;
    size_t buffer_size;

    hipGraphicsMapResources(1, &cuda_pbo_resource[nextPBO]);
    hipGraphicsResourceGetMappedPointer((void**)&d_buffer, &buffer_size, cuda_pbo_resource[nextPBO]);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);
    fillBuffer<<<gridSize, blockSize>>>(d_buffer, width, height);
    hipDeviceSynchronize();
    
	hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }


    hipGraphicsUnmapResources(1, &cuda_pbo_resource[nextPBO]);

    pboIndex = nextPBO;
}

void render(int width, int height) {
    glClear(GL_COLOR_BUFFER_BIT);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo[pboIndex]);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    GLenum err = glGetError();
    if (err != GL_NO_ERROR) {
        std::cerr << "OpenGL Error: " << err << std::endl;
    }
}