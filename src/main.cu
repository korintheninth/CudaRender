#include "hip/hip_runtime.h"
#include "libs/cudarender.h"

GLuint pbo[2];
hipGraphicsResource* cuda_pbo_resource[2]; 

int width = 1250;
int height = 720;

float3 *d_vertices;
int *d_indices;
int numTriangles;
float *depthBuffer;

triangle *d_triangles;
tile *d_tiles;

int main() {
    GLFWwindow* window = openWindow(width, height, "CUDA OpenGL Interop", NULL);
    if (!window) {
        return -1;
    }
    glfwMaximizeWindow(window);
    glfwGetWindowSize(window, &width, &height);
    hipMalloc(&depthBuffer, width * height * sizeof(float));

    createPBOs(width, height);
    glfwSetFramebufferSizeCallback(window, updateBuffersize);

    std::vector<int> indices;
    std::vector<float3> vertices;
    int numVertices;
    int numIndices;

    if (!LoadModel("objs/monkey.obj", indices, vertices, &numIndices, &numVertices)) {
        return -1;
    }
    numTriangles = numIndices / 3;
    hipMalloc(&d_triangles, numTriangles * sizeof(triangle));
    hipMalloc(&d_tiles, ((width * height) / (TILE_SIZE * TILE_SIZE)) * sizeof(tile));

    hipMalloc(&d_vertices, vertices.size() * sizeof(float3));
    hipMemcpy(d_vertices, vertices.data(), vertices.size() * sizeof(float3), hipMemcpyHostToDevice);
    
    hipMalloc(&d_indices, indices.size() * sizeof(int));
    hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(int), hipMemcpyHostToDevice);
    
	while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();
        updateContent(width, height, window);
    }

    for (int i = 0; i < 2; i++) {
        hipGraphicsUnregisterResource(cuda_pbo_resource[i]);
        glDeleteBuffers(1, &pbo[i]);
    }

    hipFree(d_indices);
    hipFree(d_vertices);
    hipFree(depthBuffer);
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
