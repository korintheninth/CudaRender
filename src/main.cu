#include "libs/cudarender.h"

GLuint pbo[2];
hipGraphicsResource* cuda_pbo_resource[2]; 
int width = 1250;
int height = 720;

int main() {

    GLFWwindow* window = openWindow(1250, 720, "CUDA OpenGL Interop", NULL);
    if (!window) {
        return -1;
    }

    createPBOs(width, height);
    glfwSetFramebufferSizeCallback(window, updatePBOsize);
    
	while (!glfwWindowShouldClose(window)) {
        updateBuffer(width, height);
        render(width, height);
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    for (int i = 0; i < 2; i++) {
        hipGraphicsUnregisterResource(cuda_pbo_resource[i]);
        glDeleteBuffers(1, &pbo[i]);
    }

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
